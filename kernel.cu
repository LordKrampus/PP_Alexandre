#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "Methods.h"


#define THREAD_LIMIT 1024
#define BLOCK_COUNT 1024


/*
* Retorna um valor arredondado da nota correspondendo a um indíce no histograma de notas
*/
__device__ int CuDetectRange(const float& nota) {
	float range;
	float fracao;

	fracao = nota - (int)nota;
	range = (int)nota;
	if (fracao >= 0.5f)
		range += 1;

	return range;
}

/*
* calcula os valores no histograma de notas
*/
__global__ void Kernel_CuCalcHist(int* hist, float* notas) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	hist[CuDetectRange(notas[index])]++;
}


__global__ void Kernel_CuAtomicCalcHist(int* hist, float* notas) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	atomicAdd(&hist[CuDetectRange(notas[index])], 1);
}


void CuCalcHistStart(float* notas, const int notas_size, int* hist, const int hist_size, int* d_hist, float* d_notas) {
	hipMalloc((void**)&d_hist, hist_size);
	hipMemcpy(d_hist, hist, hist_size, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_notas, notas_size);
	hipMemcpy(d_notas, notas, notas_size, hipMemcpyHostToDevice);
}

void CuCalcHistFinish(int* hist, const int hist_size, int* d_hist, float* d_notas) {
	hipDeviceSynchronize();
	hipMemcpy(hist, d_hist, hist_size, hipMemcpyDeviceToHost);
	
	hipFree(d_hist);
	hipFree(d_notas);
}


void CuAtomicCalcHist(float* notas, const int notas_size, int* hist, const int hist_size) {
	int* d_hist;
	float* d_notas;

	hipMalloc((void**)&d_hist, hist_size);
	hipMemcpy(d_hist, hist, hist_size, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_notas, notas_size);
	hipMemcpy(d_notas, notas, notas_size, hipMemcpyHostToDevice);

	Kernel_CuAtomicCalcHist << <1, notas_size / sizeof(float) >> > (d_hist, d_notas);

	hipDeviceSynchronize();
	hipMemcpy(hist, d_hist, hist_size, hipMemcpyDeviceToHost);

	hipFree(d_hist);
	hipFree(d_notas);
}

void CuCalcHist(float* notas, const int notas_size, int* hist, const int hist_size) {
	int* d_hist;
	float* d_notas;

	hipMalloc((void**)&d_hist, hist_size);
	hipMemcpy(d_hist, hist, hist_size, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_notas, notas_size);
	hipMemcpy(d_notas, notas, notas_size, hipMemcpyHostToDevice);

	Kernel_CuCalcHist << <1, notas_size / sizeof(float) >> > (d_hist, d_notas);

	hipDeviceSynchronize();
	hipMemcpy(hist, d_hist, hist_size, hipMemcpyDeviceToHost);

	hipFree(d_hist);
	hipFree(d_notas);
}
